/*
To Compile:
    nvcc task3.cu -o task3

To Run:
    ./task3
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//Global variable for device
__device__ char* encText;

//To calculate Time
int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference)
{
    long long int ds = finish->tv_sec - start->tv_sec;
    long long int dn = finish->tv_nsec - start->tv_nsec;

    if (dn < 0)
    {
        ds--;
        dn += 1000000000;
    }
    *difference = ds * 1000000000 + dn;

    return !(*difference > 0);
}
//Custom Encryption function to run on device
__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

//Device function to match string
__device__ int passwordMatch(char* currentEncText){
    char* check = currentEncText;
    char* match = encText;
  while(*check == *match){
    if(*check == '\0'){
        return 1;
    }
      check++;
      match++;
  }
  return 0;
    
}
//Encrypts given plain text using custom encryption
//Stores the encrypted text at global device variable
__global__ void Encrypt(){
char genRawPass[5] = "cd20";
encText = CudaCrypt(genRawPass);
//firstLetter - 'a' - 'z' (26 characters)
//secondLetter - 'a' - 'z' (26 characters)
//firstNum - '0' - '9' (10 characters)
//secondNum - '0' - '9' (10 characters)

printf("Encrypted text is: ");
printf("%c %c %c %c = %s\n", genRawPass[0],genRawPass[1],genRawPass[2],genRawPass[3], encText);
printf("Decrypting %s using Brute Force \n", encText);
}

//Cracks the the encrypted text in global variable
__global__ void crack(char * alphabet, char * numbers){

char rawPass[5];

rawPass[0] = alphabet[blockIdx.x];
rawPass[1] = alphabet[blockIdx.y];

rawPass[2] = numbers[threadIdx.x];
rawPass[3] = numbers[threadIdx.y];
rawPass[4] = '\0';

//firstLetter - 'a' - 'z' (26 characters)
//secondLetter - 'a' - 'z' (26 characters)
//firstNum - '0' - '9' (10 characters)
//secondNum - '0' - '9' (10 characters)
  
  if(passwordMatch(CudaCrypt(rawPass))){
      printf("Match Found Your Password is %s \n", rawPass);
  }

}

//Main Function
int main(int argc, char ** argv){
      //starting clock
      struct timespec start, finish;
      long long int difference;
      clock_gettime(CLOCK_MONOTONIC, &start);

	//Calls Encryption method
    Encrypt<<< 1, 1 >>>();    

    char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
    char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

    char * gpuAlphabet;
    hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26);
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    char * gpuNumbers;
    hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26);
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

    crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers );
    hipDeviceSynchronize();

    //Stopping Clock
    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &difference);
    printf("run lasted %lldns or %9.5lfs\n", difference, difference / 1000000000.0);
  
return 0;
}
#include "hip/hip_runtime.h"
/*
To Compile:
    nvcc task4.cu lodepng.cpp -o task4

To Run:
    ./task4 image.png

To Store Output:
    ./task4 image.png > CUDA_Gaussian_output.txt
*/

#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
//To calculate time
int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference)
{
  long long int ds = finish->tv_sec - start->tv_sec;
  long long int dn = finish->tv_nsec - start->tv_nsec;

  if (dn < 0)
  {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;

  return !(*difference > 0);
}
//Global Device Variable
__device__ unsigned int width;
__device__ unsigned int height;

//Device Function to get requred red color from the image
__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4);
  return image[i];
}

//Device Function to get requred green color from the image
__device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +1;
  return image[i];
}
//Device Function to get requred blue color from the image
__device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +2;
  return image[i];
}
//Device Function to get requred alpha channel from the image
__device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +3;
  return image[i];
}

//Device function to set red
__device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red){
  unsigned int i = (row * width * 4) + (col * 4);
  image[i] = red;
}

//Device Function to set Green
__device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green){
  unsigned int i = (row * width * 4) + (col * 4) +1;
  image[i] = green;
}

//Device function to set Blue
__device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue){
  unsigned int i = (row * width * 4) + (col * 4) +2;
  image[i] = blue;
}

//Device funtion to set alpha
__device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha){
  unsigned int i = (row * width * 4) + (col * 4) +3;
  image[i] = alpha;
}

//Main gunction to Blur the image in device
//Does not blur if the axis is zero to ignore the pixel at the edges
__global__ void cudaBlur(unsigned char* img_in, unsigned char* newImage){
    if(blockIdx.x == 0 || threadIdx.x == 0 ){

    }
    else{
      //Weight matrix to apply blur.
        float filter[3][3] = {
  { 1.0/16, 2.0/16, 1.0/16 },
  { 2.0/16, 4.0/16, 2.0/16 },
  { 1.0/16, 2.0/16, 1.0/16 }};
  
  //variables
    unsigned redTL,redTC, redTR;
    unsigned redL, redC, redR;
    unsigned redBL,redBC, redBR;
    unsigned newRed;

    unsigned greenTL,greenTC, greenTR;
    unsigned greenL, greenC, greenR;
    unsigned greenBL,greenBC, greenBR;
    unsigned newGreen;

    unsigned blueTL,blueTC, blueTR;
    unsigned blueL, blueC, blueR;
    unsigned blueBL,blueBC, blueBR;
    unsigned newBlue;

    int row = threadIdx.x;
    int col = blockIdx.x;

      setGreen(newImage, row, col, getGreen(img_in, row, col));
      setBlue(newImage, row, col, getBlue(img_in, row, col));
      setAlpha(newImage, row, col, 255);		

      redTL = getRed(img_in, row-1, col-1);
      redTC = getRed(img_in, row-1, col);
      redTR = getRed(img_in, row-1, col+1);

      redL = getRed(img_in, row, col-1);
      redC = getRed(img_in, row, col);
      redR = getRed(img_in, row, col+1);

      redBL = getRed(img_in, row+1, col-1);
      redBC = getRed(img_in, row+1, col);
      redBR = getRed(img_in, row+1, col+1);
      //Calculating new values for Red
      newRed = redTL*filter[0][0] + redTC*filter[0][1] + redTR*filter[0][2]
             + redL*filter[1][0]  + redC*filter[1][1]  + redR*filter[1][2]
             + redBL*filter[2][0] + redBC*filter[2][1] + redBR*filter[2][2];
      //setting new values for red
      setRed(newImage, row, col, newRed);

      greenTL = getGreen(img_in, row-1, col-1);
      greenTC = getGreen(img_in, row-1, col);
      greenTR = getGreen(img_in, row-1, col+1);

      greenL = getGreen(img_in, row, col-1);
      greenC = getGreen(img_in, row, col);
      greenR = getGreen(img_in, row, col+1);

      greenBL = getGreen(img_in, row+1, col-1);
      greenBC = getGreen(img_in, row+1, col);
      greenBR = getGreen(img_in, row+1, col+1);
      
      //Calculating new values for Green
      newGreen = greenTL*filter[0][0] + greenTC*filter[0][1] + greenTR*filter[0][2]
             + greenL*filter[1][0]  + greenC*filter[1][1]  + greenR*filter[1][2]
             + greenBL*filter[2][0] + greenBC*filter[2][1] + greenBR*filter[2][2];
      
      //Setting new values for green
      setGreen(newImage, row, col, newGreen);

      blueTL = getBlue(img_in, row-1, col-1);
      blueTC = getBlue(img_in, row-1, col);
      blueTR = getBlue(img_in, row-1, col+1);

      blueL = getBlue(img_in, row, col-1);
      blueC = getBlue(img_in, row, col);
      blueR = getBlue(img_in, row, col+1);

      blueBL = getBlue(img_in, row+1, col-1);
      blueBC = getBlue(img_in, row+1, col);
      blueBR = getBlue(img_in, row+1, col+1);
      
      //calculating new values for Blue
      newBlue = blueTL*filter[0][0] + blueTC*filter[0][1] + blueTR*filter[0][2]
             + blueL*filter[1][0]  + blueC*filter[1][1]  + blueR*filter[1][2]
             + blueBL*filter[2][0] + blueBC*filter[2][1] + blueBR*filter[2][2];
      
      //setting new values for blue
      setBlue(newImage, row, col, newBlue);        
    }
    
}

//function that sets width and height in global device variables
__global__ void setDimentions(unsigned int givenWidth, unsigned int givenHeight){
    width = givenWidth;
    height = givenHeight;
    
}

//Main function
int main(int argc, char **argv){
      for (int i = 0; i < 10; i++)
  {
    //Starting Clock
    struct timespec start, finish;
    long long int difference;
    clock_gettime(CLOCK_MONOTONIC, &start);
   
    //image variables
    unsigned char *image;
    unsigned int w;
    unsigned int h;
    //getting vthe image at runtime
    const char* filename = argv[1];
    const char* img_output = "bluredImage.png";
    unsigned char* output_raw;
    hipError_t error;
    
    //decoding image file
    lodepng_decode32_file(&image, &w, &h, filename);
    
    printf("width = %d height = %d\n", w, h);

    //calling function to set global device variables
    setDimentions<<<1,1>>>(w, h);

    unsigned char * d_image;
    const int ARRAY_BYTES = h*w*4 * sizeof(unsigned char);

    output_raw = (unsigned char *)malloc(ARRAY_BYTES);
    
    //Memory allocation and Copyting image into GPU memory
    hipMalloc((void**) &d_image, ARRAY_BYTES);
    hipMemcpy(d_image, image, ARRAY_BYTES, hipMemcpyHostToDevice);
    
    //Defining and memory allocation for output image
    unsigned char * d_out;
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    //calling main device function to blur
    cudaBlur<<<w-1, h-1>>>(d_image, d_out);
    
    //checking error
    error = hipGetLastError();
    if(error){
      fprintf(stderr, "Kernel launch returned %d %s\n", error,
        hipGetErrorString(error));
      exit(1);
    }
    //Declaration of cuda thread synchronize
    hipDeviceSynchronize();

    //Copyting output back tohost memory
    hipMemcpy(output_raw, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    //Encoding image into output file
    lodepng_encode32_file(img_output, output_raw, w, h);

    //freeing variable
    free(image);

    //Stopping Clock
    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &difference);
    printf("run no %d lasted %lldns or %9.5lfs\n", i, difference, difference / 1000000000.0);
    printf("##############################################\n");
  }

    return 0;
}